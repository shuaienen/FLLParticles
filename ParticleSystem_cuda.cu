#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
This file contains simple wrapper functions that call the CUDA kernels
*/

#include <hip/hip_runtime_api.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <GL/glew.h>
#include <cuda_gl_interop.h>

#include "hip/hip_vector_types.h"
#include "hip/hip_math_constants.h"

//cuda�Ŀ�
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "ParticleSystem_cuda.cuh"

texture<float4, 3, hipReadModeElementType> noiseTex;

// simulation parameters
__constant__ SimParams params;

// look up in 3D noise texture
__device__ float3 noise3D(float3 p)
{
	float4 n = tex3D(noiseTex, p.x, p.y, p.z);
	return make_float3(n.x, n.y, n.z);
}

// integrate particle attributes
// ��������
struct integrate_functor
{
	float deltaTime;
	float d_offset;
	float lifetime;



	__host__ __device__
		integrate_functor(float delta_time, float offset, float life_time) : deltaTime(delta_time), d_offset(offset), lifetime(life_time) {}

	template <typename Tuple>
	__device__
		void operator()(Tuple t)
	{
		volatile float4 posData = thrust::get<2>(t);

		//volatile float4 velData1 = thrust::get<4>(t);
		//volatile float4 velData2 = thrust::get<5>(t);
		volatile float4 colorData1 = thrust::get<3>(t);
		volatile float4 colorData2 = thrust::get<4>(t);

		float3 pos = make_float3(posData.x, posData.y, posData.z);

		//float3 vel1 = make_float3(velData1.x, velData1.y, velData1.z);
		//float3 vel2 = make_float3(velData2.x, velData2.y, velData2.z);
		//float3 color1 = make_float3(colorData1.x, colorData1.y, colorData1.z);
		//float3 color2 = make_float3(colorData2.x, colorData2.y, colorData2.z);

		float4 color1 = make_float4(colorData1.x, colorData1.y, colorData1.z, colorData1.w);
		float4 color2 = make_float4(colorData2.x, colorData2.y, colorData2.z, colorData2.w);

		

		//����Offset���ý���
		//float3 del_vel = vel2 - vel1;
		//float3 vel = vel1 + del_vel * d_offset;
		//float3 vel = vel2;
		//float3 del_color = color2 - color1;
		//float3 color = color1 + del_color * d_offset;
		float4 del_color = color2 - color1;
		float4 color = color1 + del_color * d_offset;

		color.w = color.w * 0.1f;
		// update particle age
		//float age = posData.w;
		//age = 0;
		//float lifetime = velData.w;

		//if (age < lifetime)
		//{
		//	age += deltaTime;
		//}
		//else
		//{
		//	age = lifetime;
		//}

		//float phase = (lifetime > 0.0) ? (age / lifetime) : 1.0; // [0, 1]
		//float fade = 1.0 - phase;
		//if (vel.y <0)
		//{
		//	fade = phase;
		//}

		// apply accelerations
		//vel += params.gravity * deltaTime;
		//vel += g * deltaTime;

		// apply procedural noise
		//float3 noise = noise3D(pos*params.noiseFreq + params.time*params.noiseSpeed);
		//vel += noise * params.noiseAmp;

		// new position = old position + velocity * deltaTime
		//pos += vel * deltaTime;

		//vel *= params.globalDamping;

		// store new position and velocity
		thrust::get<0>(t) = make_float4(pos,1);
		//thrust::get<1>(t) = make_float4(vel, lifetime);
		//thrust::get<2>(t) = make_float4(color, fade);
		thrust::get<1>(t) = color;
	}
};

struct calcDepth_functor
{
	float3 sortVector;

	__host__ __device__
		calcDepth_functor(float3 sort_vector) : sortVector(sort_vector) {}

	template <typename Tuple>
	__host__ __device__
		void operator()(Tuple t)
	{
		volatile float4 p = thrust::get<0>(t);
		float key = -dot(make_float3(p.x, p.y, p.z), sortVector); // project onto sort vector
		thrust::get<1>(t) = key;
	}
};


extern "C"
{

    hipArray *noiseArray;

    void initCuda(bool bUseGL)
    {
        if (bUseGL)
        {
            cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
        }
        else
        {
            hipSetDevice(gpuGetMaxGflopsDeviceId());
        }
    }

	// copy parameters to constant memory
    void setParameters(SimParams *hostParams)
    {
        
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
    }

    //Round a / b to nearest higher integer value
    int iDivUp(int a, int b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }

    inline float frand()
    {
        return rand() / (float) RAND_MAX;
    }

    // create 3D texture containing random values
    void createNoiseTexture(int w, int h, int d)
    {
        hipExtent size = make_hipExtent(w, h, d);
        uint elements = (uint) size.width*size.height*size.depth;

        float *volumeData = (float *)malloc(elements*4*sizeof(float));
        float *ptr = volumeData;

        for (uint i=0; i<elements; i++)
        {
            *ptr++ = frand()*2.0f-1.0f;
            *ptr++ = frand()*2.0f-1.0f;
            *ptr++ = frand()*2.0f-1.0f;
            *ptr++ = frand()*2.0f-1.0f;
        }


        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
        checkCudaErrors(hipMalloc3DArray(&noiseArray, &channelDesc, size));

        hipMemcpy3DParms copyParams = { 0 };
        copyParams.srcPtr   = make_hipPitchedPtr((void *)volumeData, size.width*sizeof(float4), size.width, size.height);
        copyParams.dstArray = noiseArray;
        copyParams.extent   = size;
        copyParams.kind     = hipMemcpyHostToDevice;
        checkCudaErrors(hipMemcpy3D(&copyParams));

        free(volumeData);

        // set texture parameters
        noiseTex.normalized = true;                      // access with normalized texture coordinates
        noiseTex.filterMode = hipFilterModeLinear;      // linear interpolation
        noiseTex.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
        noiseTex.addressMode[1] = hipAddressModeWrap;
        noiseTex.addressMode[2] = hipAddressModeWrap;

        // bind array to 3D texture
        checkCudaErrors(hipBindTextureToArray(noiseTex, noiseArray, channelDesc));
    }

    void
    integrateSystem(float4 *oldPos, float4 *newPos,
                    //float4 *newVel,
					//float4 *Vel1, float4 *Vel2, 
					float4 *newColor,
					float4 *Color1, float4 *Color2, 
                    float deltaTime, float offset,
                    int numParticles, float lifetime)
    {
        thrust::device_ptr<float4> d_newPos(newPos);
       // thrust::device_ptr<float4> d_newVel(newVel);
		thrust::device_ptr<float4> d_newColor(newColor);
        thrust::device_ptr<float4> d_oldPos(oldPos);

		//thrust::device_ptr<float4> d_Vel1(Vel1);
		//thrust::device_ptr<float4> d_Vel2(Vel2);
		thrust::device_ptr<float4> d_Color1(Color1);
		thrust::device_ptr<float4> d_Color2(Color2);

        thrust::for_each(
			//begin
            thrust::make_zip_iterator(thrust::make_tuple(d_newPos, /*d_newVel,*/ d_newColor, d_oldPos, /*d_Vel1, d_Vel2,*/ d_Color1, d_Color2)),
			//end
            thrust::make_zip_iterator(thrust::make_tuple(d_newPos+numParticles, /*d_newVel+numParticles,*/ d_newColor+numParticles, d_oldPos+numParticles, /*d_Vel1+numParticles, d_Vel2+numParticles,*/ d_Color1+numParticles, d_Color2+numParticles)),
			//operation
            integrate_functor(deltaTime, offset, lifetime));
    }

	// ����λ�ú�sortVector��half_vector�����õ�����numParticles������keys��index
    void
    calcDepth(float4  *pos,
              float   *keys,        // output keyΪ����pos��half_vector dot��Ľ��
              uint    *indices,     // output
              float3   sortVector,
              int      numParticles)
    {
        thrust::device_ptr<float4> d_pos(pos);
        thrust::device_ptr<float> d_keys(keys);
        thrust::device_ptr<uint> d_indices(indices);

        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos, d_keys)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos+numParticles, d_keys+numParticles)),
            calcDepth_functor(sortVector));

        thrust::sequence(d_indices, d_indices + numParticles);
    }

	//����sortKeys����indices
    void sortParticles(float *sortKeys, uint *indices, uint numParticles)
    {
        thrust::sort_by_key(thrust::device_ptr<float>(sortKeys),
                            thrust::device_ptr<float>(sortKeys + numParticles),
                            thrust::device_ptr<uint>(indices));
    }

}   // extern "C"
